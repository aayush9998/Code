#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o crackpwd_2alph2digt crackpwd_2alpha_2digit.cu


     To Run:
     ./crackpwd_2alph2digt > resultscuda_2alp2dig.txt

  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is
  identical to the plain text password string stored in the program.
  Otherwise,it returns 0.
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char password1[] = "DV72";
  char password2[] = "ET21";
  char password3[] = "IR24";
  char password4[] = "SD49";

  char *a = attempt;
  char *y = attempt;
  char *u = attempt;
  char *s = attempt;
  char *pass1 = password1;
  char *pass2 = password2;
  char *pass3 = password3;
  char *pass4 = password4;

  while(*a == *pass1) {
   if(*a == '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    a++;
    pass1++;
  }
    
  while(*y == *pass2) {
   if(*y == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    y++;
    pass2++;
  }

  while(*u == *pass3) {
   if(*u == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    u++;
    pass3++;
  }

  while(*s == *pass4) {
   if(*s == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    s++;
    pass4++;
  }
  return 0;

}
/****************************************************************************
  The kernel function assume that there will be only one thread and uses
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char h,t;
 
  char password[5];
  password[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(h='0'; h<='9'; h++){
      for(t='0'; t<='9'; t++){
            password[2] = h;
            password[3] = t;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}




